#include "hip/hip_runtime.h"
//a##################################################################
//a##
//a##                    CUDA-VTI-RTM-ADCIGs
//a##
//a##----------------------------------------------------------------
//a## Features:
//a##    Read initial models & shots derive Migrations and ADCIGs, 
//a## use poynting vector method to calculate angle of reflection.
//a## This is a CUDA code, initial code comes from "/Madagascar/user
//a## /pyang/Mgpu3dfd.cu". That code don't include boundary condition
//a## and it's a 3-D forward.
//a##----------------------------------------------------------------
//a##
//a##
//a##              | npml |mm|    nx    |mm| npml |
//a##           -- 0------------------------------> nx+2*mm+2*npml
//a##         npml |             npml             |
//a##           -- |   ------------------------   |
//a##           mm |   |          mm          |   |
//a##           -- |   |   ----------------   |   |
//a##              |   |   |              |   |   |
//a##              |   |   |              |   |   |
//a##           nz |   |   |     ved      |   |   |
//a##              |   |   |              |   |   |
//a##              |   |   |              |   |   |
//a##           -- |   |   ----------------   |   |
//a##           mm |   |                      |   |
//a##           -- |   ------------------------   |
//a##         npml |                              |
//a##           -- |-------------------------------
//a##       nz+2*mm+2*npml         
//a##
//a##---------------------------------------------------------------
//a## Ps: some of function you can search in Madagascar/user/pyang
//a##
//a##
//a##
//a##---------------------------------------------------------------
//a##                                            Rong Tao
//a##################################################################
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define PI 	3.141592653

#define BlockSize1 32// tile size in 1st-axis
#define BlockSize2 32// tile size in 2nd-axis

#define mm      4    // half of the order in space
#define npml     50   // absorbing boundry condition wield

//a################################################################################
__global__ void cuda_cal_c(float *c)
/*< get source >*/
{
  int id=threadIdx.x+blockIdx.x*blockDim.x;
    if(id<1){
       if(mm==2){
        c[0]=1.125;
        c[1]=-0.04166667;
	}else if(mm==3){
	 c[0]=1.1718750;
        c[1]=-0.065104167;
        c[2]=0.0046875;
	}else if(mm==4){
	 c[0]=1.196289;
        c[1]=-0.0797526;
        c[2]=0.009570313;
        c[3]=-0.0006975447;
	}else if(mm==5){
	 c[0]=1.211243;
        c[1]=-0.08972168;
        c[2]=0.01384277;
        c[3]=-0.00176566;
        c[4]=0.0001186795;
	}else if(mm==6){
        c[0]=1.2213364;
        c[1]=-0.096931458;
        c[2]=0.017447662;
        c[3]=-0.0029672895;
        c[4]=0.0003590054;
        c[5]=-0.000021847812;
   	}else if(mm==7){
        c[0]=1.2286062;
        c[1]=-0.10238385;
        c[2]=0.020476770;
        c[3]=-0.0041789327;
        c[4]=0.00068945355;
        c[5]=-0.000076922503;
        c[6]=0.0000042365148;
       }else if(mm==8){
        c[0]=1.2340911;
        c[1]=-0.10664985;
        c[2]=0.023036367;
        c[3]=-0.0053423856;
        c[4]=0.0010772712;
        c[5]=-0.00016641888;
        c[6]=0.000017021711;
        c[7]=-0.00000085234642;
   	}  
     }           
}
//a################################################################################
void check_gpu_error (const char *msg) 
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) { 
	printf("Cuda error: %s: %s\n", msg, hipGetErrorString (err)); 
	exit(0);   
    }
}
//a################################################################################
__constant__ float stencil[mm+1]={-205.0/72.0,8.0/5.0,-1.0/5.0,8.0/315.0,-1.0/560.0};
//a################################################################################
__global__ void cuda_ricker_wavelet(float *wlt, float fm, float dt, int nt)
/*< generate ricker wavelet with time deley >*/
{
	int it=threadIdx.x+blockDim.x*blockIdx.x;
    	if (it<nt){
	  float tmp = PI*fm*fabsf(it*dt-1.0/fm);//delay the wavelet to exhibit all waveform
	  tmp *=tmp;
	  wlt[it]= (1.0-2.0*tmp)*expf(-tmp);// ricker wavelet at time: t=nt*dt
	}
}
//a################################################################################
__global__ void cuda_set_s(int *szx, int fsz, int fsx, int dsz, int dsx, int ns, int nz, int nx)
/*< set the positions of sources  in whole domain >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
	int nnz=nz+2*mm+2*npml;
    	if (id<ns) szx[id]=(fsz+id*dsz+mm+npml)+nnz*(fsx+id*dsx+mm+npml);
}
//a################################################################################
__global__ void cuda_set_g(int *gzx, int ng, int nz, int nx)
/*< set the positions of  geophones in whole domain >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
	int nnz=nz+2*mm+2*npml;
       int ix=id%nx;
    	if (id<ng) gzx[id]=(mm+npml)+nnz*(ix*1+mm+npml);
}
//a################################################################################
__global__ void cuda_trans_x2tx(float *x, float *tx, int it, int nt, int ng, bool flag)
/*< set the positions of  geophones in whole domain >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
    	if (id<ng){
          if(flag){
             tx[it+id*nt]=x[id];
           }else{
             x[id]=tx[it+id*nt];
            }
        }
}
//a################################################################################
__global__ void cuda_absorb_bndr(float *p1,float *p2,float *p3,float *p4,int nz,int nx,float qp)
/*< absorb boundry condition >*/
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id;
	int nnz=nz+2*mm+2*npml;

          id=iz+ix*nnz;
            /*< left & right (0<x<nx) >*/
             if ( ix < npml ){
               p1[id]=( qp*pow((npml-ix)/(1.0*npml),2) + 1 )*p1[id];
               p2[id]=( qp*pow((npml-ix)/(1.0*npml),2) + 1 )*p2[id];
               p3[id]=( qp*pow((npml-ix)/(1.0*npml),2) + 1 )*p3[id];
               p4[id]=( qp*pow((npml-ix)/(1.0*npml),2) + 1 )*p4[id];
             }else if ( ix >= 2*mm + npml + nx ){
               p1[id]=( qp*pow((ix-2*mm-npml-nx)/(1.0*npml),2) + 1 )*p1[id];
               p2[id]=( qp*pow((ix-2*mm-npml-nx)/(1.0*npml),2) + 1 )*p2[id];
               p3[id]=( qp*pow((ix-2*mm-npml-nx)/(1.0*npml),2) + 1 )*p3[id];
               p4[id]=( qp*pow((ix-2*mm-npml-nx)/(1.0*npml),2) + 1 )*p4[id];
              }
            /*< up & down (0<z<nz) >*/
             if ( iz < npml ){
               p1[id]=( qp*pow((npml-iz)/(1.0*npml),2) + 1 )*p1[id];
               p2[id]=( qp*pow((npml-iz)/(1.0*npml),2) + 1 )*p2[id];
               p3[id]=( qp*pow((npml-iz)/(1.0*npml),2) + 1 )*p3[id];
               p4[id]=( qp*pow((npml-iz)/(1.0*npml),2) + 1 )*p4[id];
             }else if ( iz >= 2*mm + npml + nz ){
               p1[id]=( qp*pow((iz-2*mm-npml-nz)/(1.0*npml),2) + 1 )*p1[id]; 
               p2[id]=( qp*pow((iz-2*mm-npml-nz)/(1.0*npml),2) + 1 )*p2[id]; 
               p3[id]=( qp*pow((iz-2*mm-npml-nz)/(1.0*npml),2) + 1 )*p3[id]; 
               p4[id]=( qp*pow((iz-2*mm-npml-nz)/(1.0*npml),2) + 1 )*p4[id];   
               }
}
//a################################################################################
//__global__ void cuda_initial_PML(float *coff1, float *coff2, int nx, int nz, float dx, float dz, float dt, float vmax)
/*< PML boundry condition >*/
/*{
    int id=threadIdx.x+blockIdx.x*blockDim.x;

    float d0=3.0*vmax*log(100000.0)/(2.0*npml*(dx+dz)/2);
    int iz=id%(nz+2*mm+2*npml);
    int ix=id/(nz+2*mm+2*npml);

   if(id<(nx+2*npml*2*mm)*(nz+2*npml*2*mm))
   {
      if(iz<=npml){
            coff1[id]=1/(1+(dt*d0*pow((npml-0.5-iz)/npml,2))/2);
            coff2[id]=coff1[id]*(1-(dt*d0*pow((npml-0.5-iz)/npml,2))/2);
      }else if(iz>=nz+2*mm+npml){
            coff1[id]=1/(1+(dt*d0*pow((0.5+iz-nz-2*mm-npml)/npml,2))/2);
	     coff2[id]=coff1[id]*(1-(dt*d0*pow((0.5+iz-nz-2*mm-npml)/npml,2))/2);
      }if(ix<=npml&&(ix<=iz)&&ix<=(nz+2*mm+2*npml-iz)){
            coff1[id]=1/(1+(dt*d0*pow((npml-0.5-ix)/npml,2))/2);
            coff2[id]=coff1[id]*(1-(dt*d0*pow((npml-0.5-ix)/npml,2))/2);
      }else if((ix>=nx+2*mm+npml)&&(ix>=iz+nx-nz)&&(iz>=nx+2*mm+2*npml-ix)){
            coff1[id]=1/(1+(dt*d0*pow((0.5+ix-nx-2*mm-npml)/npml,2))/2);
	     coff2[id]=coff1[id]*(1-(dt*d0*pow((0.5+ix-nx-2*mm-npml)/npml,2))/2);
      }if(ix>=npml&&ix<=(npml+nx+2*mm)&&iz>=npml&&iz<=(npml+nz+2*mm)){
            coff1[id]=1.0;
	     coff2[id]=1.0;
      }
   }        
}*/
//a################################################################################
__global__ void cuda_record(float *p, float *seis, int *gx, int ng, bool flag)//++++++++++++
/*< record the seismogram at time it >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
    	if (id<ng) {
           if(flag){
                seis[id]=p[gx[id]];
            }else{
                p[gx[id]]=seis[id];
             }
        }
}
//a################################################################################
__global__ void cuda_add_source(bool add, float *p, float *source, int *szx, int ns)
/*< add/subtract sources: length of source[]=ns, index stored in szxy[] >*/
{
  int id=threadIdx.x+blockIdx.x*blockDim.x;

  if(id<ns){
    if(add){
      p[szx[id]]+=source[id];
    }else{
      p[szx[id]]-=source[id];
    }
  }
}
//a################################################################################
__global__ void cuda_step_fd2d(float *p0, float *p1, float *q0, float *q1, float *vv, float *vx, float *vn, 
                               float _dz2, float _dx2,int nz, int nx, 
                               bool forward, int *szx, int r, int R)
/*< step forward: 3-D FD, order=8 >*/
{
    bool validr = true;
    bool validw = true;
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix
    const int ltid1 = threadIdx.x;//ithreadz
    const int ltid2 = threadIdx.y;//ithreadx
    const int work1 = blockDim.x;//nblockz
    const int work2 = blockDim.y;//nblockx
    __shared__ float tile[BlockSize2 + 2 * mm][BlockSize1 + 2 * mm];
    __shared__ float tile2[BlockSize2 + 2 * mm][BlockSize1 + 2 * mm];

    float vvv, vvx, vvn;

    const int stride2 = nz + 2 * mm + 2 * npml;
    int inIndex = 0;
    int outIndex = 0;

    // Advance inputIndex to start of inner volume
    inIndex += (mm ) * stride2 + mm ;
    // Advance inputIndex to target element
    inIndex += ix * stride2 + iz;

    float current, current2;
    const int t1 = ltid1 + mm;
    const int t2 = ltid2 + mm;
    // Check in bounds
    if ((iz >= nz + mm + 2*npml) ||(ix >= nx + mm + 2*npml)) validr = false;
    if ((iz >= nz + 2*npml) ||(ix >= nx + 2*npml)) validw = false;

    if (validr) {current = p1[inIndex];current2 = q1[inIndex];}

    outIndex = inIndex;
    __syncthreads();

    if (ltid2 < mm){

       tile[ltid2][t1]                  = p1[outIndex - mm * stride2];
       tile[ltid2 + work2 + mm][t1]     = p1[outIndex + work2 * stride2];
       tile2[ltid2][t1]                  = q1[outIndex - mm * stride2];
       tile2[ltid2 + work2 + mm][t1]     = q1[outIndex + work2 * stride2];

    }if (ltid1 < mm){// Halo left & right

       tile[t2][ltid1]                  = p1[outIndex - mm];
       tile[t2][ltid1 + work1 + mm]     = p1[outIndex + work1];
       tile2[t2][ltid1]                  = q1[outIndex - mm];
       tile2[t2][ltid1 + work1 + mm]     = q1[outIndex + work1];
     }

    tile[t2][t1] = current;
    tile2[t2][t1] = current2;
    __syncthreads();

   // Compute the output value
    float c2, c3;
    c2=stencil[0]*current;
    c3=stencil[0]*current2;        

    for (int i=1; i <= mm ; i++){
	c2 +=stencil[i]*(tile[t2-i][t1]+ tile[t2+i][t1]);//x
       c3 +=stencil[i]*(tile2[t2][t1-i]+ tile2[t2][t1+i]);//z
     }
    c2*=_dx2;
    c3*=_dz2;	
    if (validw){
     if(!forward){
          vvv=vv[outIndex];
          vvx=vx[outIndex];
          vvn=vn[outIndex];
     }else{

       int iix=outIndex/stride2;
       int iiz=outIndex%stride2;

       int sx=*szx/stride2;
       int sz=*szx%stride2;

       int d=(int)sqrtf( ((iix-sx)*(iix-sx)) + ((iiz-sz)*(iiz-sz)) );

           if(d<=r){
               vvv=vv[outIndex];
               vvx=vv[outIndex];
               vvn=vv[outIndex];
           }else if(d>r&&d<=R){
               vvv=vv[outIndex];
               vvx=vv[outIndex]+(vx[outIndex]-vv[outIndex])/2.0*(1+cos( PI/(R-r)*(d-r) +PI ));
               vvn=vv[outIndex]+(vn[outIndex]-vv[outIndex])/2.0*(1+cos( PI/(R-r)*(d-r) +PI ));
           }else{
               vvv=vv[outIndex];
               vvx=vx[outIndex];
               vvn=vn[outIndex];
            }
      }

       p0[outIndex]=2.0*p1[outIndex]-p0[outIndex]
                   +vvv*(c3)+vvx*(c2);

       q0[outIndex]=2.0*q1[outIndex]-q0[outIndex]
                   +vvv*(c3)+vvn*(c2);

    }
}
//a################################################################################
//__global__ void cuda_iso_source_ring(float *vv, float *vx, float *vn, int nx, int nz, int *szx, int r, int R)
///*< Isotropic source ring >*/
//{
//	int id=threadIdx.x+blockDim.x*blockIdx.x;
//
//       int nnx=nx+2*mm+2*npml;
//       int nnz=nz+2*mm+2*npml;
//
//       int ix=id/nnz;
//       int iz=id%nnz;
//
//       int sx=*szx/nnz;
//       int sz=*szx%nnz;
//
//       int d=(int)sqrtf( ((ix-sx)*(ix-sx)) + ((iz-sz)*(iz-sz)) );
//
//    	if (id<nnx*nnz){
//           if(d<=r){
//               vx[id]=vv[id];
//               vn[id]=vv[id];
//            }else if(d>r&&d<=R){
//               vx[id]=vv[id]+(vx[id]-vv[id])/2.0*(1+cos( PI/(R-r)*(d-r) +PI ));
//               vn[id]=vv[id]+(vn[id]-vv[id])/2.0*(1+cos( PI/(R-r)*(d-r) +PI ));
//             }else{}
//        }
//
//}
//a################################################################################
void velocity_transform(float *v0, float*vv, float*vx, float*vn, float dt, int nz, int nx, float *vmax, float *vmin, float *vmute)
 /*< velocit2 transform: vv=v0*dt; vv<--vv^2 >*/
{
  int i1, i2, nnz, nnx;
  float tmp;

  nnz=nz+2*mm+2*npml;
  nnx=nx+2*mm+2*npml;
  *vmax=v0[0];
  *vmin=v0[0];
  *vmute=v0[0]*sqrtf(1+2*0.3);
  // inner zone
    for(i2=0; i2<nx; i2++){//x
      for(i1=0; i1<nz; i1++){//z
       if(*vmax<v0[i1+nz*i2])*vmax=v0[i1+nz*i2];
       if(*vmin>v0[i1+nz*i2])*vmin=v0[i1+nz*i2];
	tmp=v0[i1+nz*i2]*dt;
	vv[(i1+mm+npml)+nnz*(i2+mm+npml)]=tmp*tmp;
	vx[(i1+mm+npml)+nnz*(i2+mm+npml)]=tmp*tmp*(1+2*0.3);
	vn[(i1+mm+npml)+nnz*(i2+mm+npml)]=tmp*tmp*(1+2*0.2);
      }
    }
    //top & down 
	for(i2=0; i2<nnx; i2++){//x
	    for (i1=0; i1<mm+npml; i1++){//z
		vv[i1+nnz*i2]=vv[mm+npml+nnz*i2];
		vv[(nnz-i1-1)+nnz*i2]=vv[(nnz-mm-npml-1)+nnz*i2];
		vx[i1+nnz*i2]=vx[mm+npml+nnz*i2];
		vx[(nnz-i1-1)+nnz*i2]=vx[(nnz-mm-npml-1)+nnz*i2];
		vn[i1+nnz*i2]=vn[mm+npml+nnz*i2];
		vn[(nnz-i1-1)+nnz*i2]=vn[(nnz-mm-npml-1)+nnz*i2];
	    }
	}
    //left & right
	for(i2=0; i2<mm+npml; i2++){//x
	    for (i1=0; i1<nnz; i1++){//z
		vv[i1+nnz*i2]=vv[i1+nnz*(mm+npml)];
		vv[i1+nnz*(nnx-i2-1)]=vv[i1+nnz*(nnx-mm-npml-1)];
		vx[i1+nnz*i2]=vx[i1+nnz*(mm+npml)];
		vx[i1+nnz*(nnx-i2-1)]=vx[i1+nnz*(nnx-mm-npml-1)];
		vn[i1+nnz*i2]=vn[i1+nnz*(mm+npml)];
		vn[i1+nnz*(nnx-i2-1)]=vn[i1+nnz*(nnx-mm-npml-1)];
	    }
	}
}
//a################################################################################
void window3d(float *a, float *b, int nz, int nx)
/*< window a 3d subvolume >*/
{
	int i1, i2, nnz;
	nnz=nz+2*mm+ 2*npml;//z
	
	for(i2=0; i2<nx; i2++)
	for(i1=0; i1<nz; i1++)
	{
          a[i1+nz*i2]=b[(i1+mm+npml)+nnz*(i2+mm+npml)];
	}
}
//a################################################################################
__global__ void cuda_set_cooLR(int *left, int *right, int nx, int nz)
/*< set the positions of  left & right in whole domain >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
	int nnz=nz+2*mm+2*npml;
    	if (id<nz){
            left[id]=(nnz+1)*(mm+npml)+id;
           right[id]=nnz*(mm+npml+nx)+mm+npml+id;
        }
}
//a################################################################################
__global__ void cuda_set_cooUD(int *up, int *down, int nx, int nz)
/*< set the positions of  up & down in whole domain >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
	int nnz=nz+2*mm+2*npml;
    	if (id<nx){
             up[id]=(nnz+1)*(mm+npml)+id*nnz;
           down[id]=(nnz+1)*(mm+npml)+id*nnz+nz;
        }
}
//a#############################################################################################
__global__ void cuda_s_bndr(float *s_p_bndr, float *s_q_bndr, float *p, float *q, 
                            int *left, int *right, int *up, int *down, int nz, int nx, bool write)
/*< write boundaries out or read them into wavefield variables p>*/
{
	int id=threadIdx.x+blockIdx.x*blockDim.x;
	if(write){
		if(id<nz){ /* left  boundary */
                    s_p_bndr[id]=p[left[id]]; 
                    s_q_bndr[id]=q[left[id]];         
		}else if((id>=nz)&&(id<(2*nz))){ /* right boundary */
                    s_p_bndr[id]=p[right[id-nz]]; 
                    s_q_bndr[id]=q[right[id-nz]];  
		}else if(id>=(2*nz)&&(id<(2*nz+nx))){  /* up    boundary */
                    s_p_bndr[id]=p[up[id-2*nz]];  
                    s_q_bndr[id]=q[up[id-2*nz]];  
		}else if(id>=(2*nz+nx)&&id<(2*nz+2*nx)){ /* down boundary */
                    s_p_bndr[id]=p[down[id-2*nz-nx]];
                    s_q_bndr[id]=q[down[id-2*nz-nx]];
                }
	}else{
		if(id<nz){ /* left  boundary */
                    p[left[id]]=s_p_bndr[id]; 
                    q[left[id]]=s_q_bndr[id];         
		}else if((id>=nz)&&(id<(2*nz))){ /* right boundary */
                    p[right[id-nz]]=s_p_bndr[id]; 
                    q[right[id-nz]]=s_q_bndr[id];  
		}else if(id>=(2*nz)&&(id<(2*nz+nx))){  /* up    boundary */
                    p[up[id-2*nz]]=s_p_bndr[id];  
                    q[up[id-2*nz]]=s_q_bndr[id];  
		}else if(id>=(2*nz+nx)&&id<(2*nz+2*nx)){ /* down boundary */
                    p[down[id-2*nz-nx]]=s_p_bndr[id];
                    q[down[id-2*nz-nx]]=s_q_bndr[id];
                }
        }
}
//a################################################################################
__global__ void cuda_cal_corr(float *mig, float *s, float *g, int nx, int nz)
/*< correlation imaging condition >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
    	if (id<(nx+2*mm+2*npml)*(nz+2*mm+2*npml)){
              mig[id]+=s[id]*g[id];
        }
}
//a################################################################################
__global__ void cuda_cal_illum_matrix(float *illum, float *wave, int nx, int nz)
/*< illumination matrix >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
    	if (id<(nx+2*mm+2*npml)*(nz+2*mm+2*npml)){
              illum[id]+=wave[id]*wave[id];
        }
}
//a################################################################################
__global__ void cuda_illumination(float *mig_ns, float *mig_is, float *illum_ns, float *illum_is, int nx, int nz)
/*< illumination matrix >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
    	if (id<(nx+2*mm+2*npml)*(nz+2*mm+2*npml)){
              mig_is[id]/=illum_is[id];
              mig_ns[id]+=mig_is[id];
              illum_ns[id]+=illum_is[id];
        }
}
//a################################################################################
__global__ void cuda_mute_direct(float *p, int nx, int nz, float dx, float dz, int nt, float dt, float fm, 
                                 float vmute, int *coo_zx, int tt)
/*< illumination matrix >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;

       int ix=id/nt;
       int it=id%nt;

       int sx=*coo_zx/(nz+2*mm+2*npml)-mm-npml;
       int sz=*coo_zx%(nz+2*mm+2*npml)-mm-npml;

       int t0=(int)sqrtf((dx*(ix-sx)*dx*(ix-sx))+(dz*sz*dz*sz));
       int t1=(int)(t0/vmute/dt);
       int t2=(int)(2.0/(dt*fm));

    	if (id<nx*nt){
          if(it <= t1+t2+tt){
              p[id]=0.0;     
           }else{}
        }

}
//a################################################################################
__global__ void cuda_mut_v(float *s_vv, float *s_vx, float *s_vn, float *mut_vv, float *mut_vx, float *mut_vn, 
                           int nx, int nz)
/*< copy velocity to mute velocity >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;

       int nnx=nx+2*mm+2*npml;
       int nnz=nz+2*mm+2*npml;

    	if (id<nnx*nnz){
           mut_vv[id]=s_vv[0];
           mut_vx[id]=s_vx[0];
           mut_vn[id]=s_vn[0];
        }
}
//a################################################################################
__global__ void cuda_difference(float *p1, float *p2, int nx, int nt)
/*< get wavefiled(x-t) difference >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;

    	if (id<nx*nt) p1[id]-=p2[id];
}
//a################################################################################
__global__ void cuda_poynting_adcigs(float *adcigs_Dev, float *s_p0, float *s_q0, float *g_p0, float *g_q0, 
                                     int nx, int nz, int na, float _dx2, float _dz2, float *illum_is, int *angle_count_Dev, 
                                     float *c)
/*< poynting vector get ADCIGs >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;

       int nnz=nz+2*mm+2*npml;
       int nnx=nx+2*mm+2*npml;

       int iz=id%nnz-mm-npml;
       int ix=id/nnz-mm-npml;

       float Ssx, Ssz, Sgx, Sgz, tmp, s_u=0.0, s_w=0.0, g_u=0.0, g_w=0.0;

       s_u=stencil[0]*s_p0[id]*sqrtf(_dx2);
       s_w=stencil[0]*s_q0[id]*sqrtf(_dz2);
       g_u=stencil[0]*g_p0[id]*sqrtf(_dx2);
       g_w=stencil[0]*g_q0[id]*sqrtf(_dz2);

       for(int im=1; im<=mm; im++){
            s_u+=stencil[im]*(s_p0[id+im*nnz]+s_p0[id-im*nnz])*sqrtf(_dx2);
            s_w+=stencil[im]*(s_p0[id+im]    +s_p0[id-im])      *sqrtf(_dz2);
            g_u+=stencil[im]*(g_p0[id+im*nnz]+g_p0[id-im*nnz])*sqrtf(_dx2);
            g_w+=stencil[im]*(g_p0[id+im]    +g_p0[id-im])      *sqrtf(_dz2);
        }

       Ssx=-s_p0[id]*s_u;
       Ssz=-s_q0[id]*s_w;
       Sgx= g_p0[id]*g_u;
       Sgz= g_q0[id]*g_w;

       float b1=Ssz*Ssz+Ssx*Ssx;
       float b2=Sgz*Sgz+Sgx*Sgx;

       float a= 0.5*acosf( (Ssx*Sgx+Ssz*Sgz) / (sqrtf(b1*b2)*(1+0.0)) );

       int ia=(int)(  a*180/PI  );

    	if (((ia>=0)&&(ia<na))&&iz>-1&&ix>-1&&id<nnz*nnx&&(iz+nz*ia+nz*na*ix)<nz*na*nx){
              angle_count_Dev[ia]++;
              tmp=s_p0[id]*g_p0[id];//+s_q0[id]*g_q0[id];
              //tmp=tmp*expf(-(a-ia)*(a-ia)/ ( 2.0/9.0 ) );
              adcigs_Dev[iz+nz*ia+nz*na*ix]+=tmp/illum_is[id];
        }
}
//a################################################################################
__global__ void cuda_smooth_adcigs(float *adcigs, int nx, int nz, int na, int nsmooth)
/*< poynting vector get ADCIGs >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;

       int iz=id%nz;
       int ix=id/nz;

    if (id<nz*nx)
      for(int in=0; in<nsmooth; in++){
         for(int ia=0; ia<na; ia++){
            if((iz+nz*ia+nz*na*ix)<nz*na*nx)
              if(ia==0){
                adcigs[iz+nz*ia+nz*na*ix]=(adcigs[iz+nz*na*ix]+adcigs[iz+nz+nz*na*ix])/2.0;
              }else if(ia==na-1){
                adcigs[iz+nz*ia+nz*na*ix]=(adcigs[iz+nz*(na-1)+nz*na*ix]+adcigs[iz+nz*(na-2)+nz*na*ix])/2.0;
              }else{
                adcigs[iz+nz*ia+nz*na*ix]=(adcigs[iz+nz*(ia-1)+nz*na*ix]+adcigs[iz+nz*(ia+1)+nz*na*ix]+adcigs[iz+nz*ia+nz*na*ix])/3.0;
               }
          }
       }  
}
//a################################################################################
//__global__ void cuda_uw(float *p, float *q, float _dz2, float _dx2,int nz, int nx, float dt, float *u, float *w)
///*< get zhidian velocity u,w >*/
//{
//	int id=threadIdx.x+blockDim.x*blockIdx.x;
//
//       int nnz=nz+2*mm+2*npml;
//       int nnx=nx+2*mm+2*npml;
//       float diffx, diffz;
//
//       if(id<nnx*nnz){
//           diffx=1.125*(p[id+nnz]-p[id])-0.0416666666667*(p[id+2*nnz]-p[id+nnz]);
//           diffz=1.125*(q[id+1]-q[id])-0.0416666666667*(q[id+2]-q[id+1]);
//           u[id]+=1000*dt*sqrtf(_dx2)*diffx;
//           w[id]+=1000*dt*sqrtf(_dz2)*diffz;
//        }
//}
//a################################################################################
void laplace_filter(int adj, int nz, int nx, float *in, float *out)
/*< linear operator, come from Madagascar Mlaplac2>*/
{
    int iz,ix,j;
    for (j=0;j<nx*nz;j++) out[j]=0.0;
    for (ix=0; ix < nx; ix++) {
	for (iz=0; iz < nz; iz++) {
	    j = iz+ix*nz;
	    if (iz > 0) {
		if (adj) {
		    out[j-1] -= in[j];
		    out[j]   += in[j];
		} else {
		    out[j] += in[j] - in[j-1];
		}
	    }
	    if (iz < nz-1) {
		if (adj) {
		    out[j+1] -= in[j];
		    out[j]   += in[j];
		} else {
		    out[j] += in[j] - in[j+1];
		}
	    }

	    if (ix > 0) {
		if (adj) {
		    out[j-nz] -= in[j];
		    out[j]    += in[j];
		} else {
		    out[j] += in[j] - in[j-nz];
		}
	    }
	    if (ix < nx-1) {
		if (adj) {
		    out[j+nz] -= in[j];
		    out[j]    += in[j];
		} else {
		    out[j] += in[j] - in[j+nz];
		}
	    }
	}
    }
}
//a################################################################################
//a###                                                                         ####
//a###                             Main Function                               ####
//a###                                                                         ####
//a################################################################################
int main(int argc, char* argv[])
{
	int nz, nx, nnz, nnx, ns, nt, kt, it, is, fsz, fsx,  dsz, dsx, ng, i, na, r, R;
	int *szx, *gzx, *coo_left, *coo_right, *coo_up, *coo_down, *angle_count_Dev, *angle_count_Hos;
	float dz, dx,  fm, dt, _dz2, _dx2, vmax=0.0, vmin=0.0, vmute=0.0;
	float *temp, *v0, *vv, *vx, *vn, *s_wlt, *s_vv, *s_vx, *s_vn, *s_p0, *s_p1, *s_q0, *s_q1, *g_p0, *g_p1, *g_q0, *g_q1, *ptr;
       float *cal_it_Dev, *cal_nt_Dev, *cal_nt_Hos, *obs_it_Dev, *obs_nt_Dev, *obs_nt_Hos, *s_p_bndr, *s_q_bndr;
       float *c;
       float *mig_is, *mig_ns, *illum_is, *illum_ns;
       float *mut_vv, *mut_vx, *mut_vn, *mut_p0, *mut_p1, *mut_q0, *mut_q1, *mut_nt_Dev;
       float *adcigs_Dev, *adcigs_Hos;
       bool flag_snap, flag_laplace, flag_adcigs_smooth;
//a##################################################
	char     FNvel[250]={"vel_600_300.dat"};
       char    FNsnap[250]={"snap.dat"};
       char FNshotcal[250]={"shot_cal.dat"};
       char FNshotobs[250]={"shot_cal.dat"};
       char   FNmigis[250]={"mig_is.dat"};
       char   FNmigns[250]={"mig_ns.dat"};
       char FNillumis[250]={"illum_is.dat"};
       char FNillumns[250]={"illum_ns.dat"};
       char  FNadcigs[250]={"adcigs.dat"};
//a##################################################
             flag_snap=true;
          flag_laplace=true;
    flag_adcigs_smooth=true;
//a##################################################
       r=10; R=20;
//a##################################################
       fm=20;     

    	nx=300;   dx=10;
    	nz=300;   dz=10;
    	
   	nt=2501;   kt=50;    dt=0.001;

   	ns=10;
       fsx=nx/ns/2;    dsx=nx/ns;
       fsz=1;    dsz=0;

       na=70;

//a##################################################
       FILE *fpvel, *fpsnap, *fpshotcal, *fpshotobs, *fpmigis, *fpmigns, *fpillumis, *fpillumns, *fpadcigs;
       if((fpvel=fopen(FNvel,"rb"))==NULL) printf("ERROR:open %s error!\n",FNvel);
       if(flag_snap) fpsnap=fopen(FNsnap,"wb");
       fpshotcal=fopen(FNshotcal,"wb");
       fpshotobs=fopen(FNshotobs,"rb");
         fpmigis=fopen(FNmigis,"wb");
         fpmigns=fopen(FNmigns,"wb");
       fpillumis=fopen(FNillumis,"wb");
       fpillumns=fopen(FNillumns,"wb");
        fpadcigs=fopen(FNadcigs,"wb");

//a##################################################
	_dz2=1.0/(dz*dz);
	_dx2=1.0/(dx*dx);
	nnz=nz+2*mm+2*npml;
	nnx=nx+2*mm+2*npml;
       ng=nx;
//a##################################################
    	v0=(float*)malloc(nz*nx*sizeof(float));
    	temp=(float*)malloc(nz*nx*sizeof(float));
    	vv=(float*)malloc(nnz*nnx*sizeof(float));
    	vx=(float*)malloc(nnz*nnx*sizeof(float));
    	vn=(float*)malloc(nnz*nnx*sizeof(float));
    	cal_nt_Hos=(float*)malloc(ng*nt*sizeof(float));
    	obs_nt_Hos=(float*)malloc(ng*nt*sizeof(float));
    	adcigs_Hos=(float*)malloc(nz*na*nx*sizeof(float));
    	angle_count_Hos=(int*)malloc(na*sizeof(int));
	fread(v0, sizeof(float), nz*nx, fpvel);
	velocity_transform(v0, vv, vx, vn, dt, nz, nx, &vmax, &vmin, &vmute);
//a##################################################
       printf("###############################\n");
       printf("##  vmin=%.2f, vmax=%.2f\n",vmin,vmax);
       printf("###############################\n");
//a##################################################
    	hipSetDevice(0);// initialize device, default device=0;
	check_gpu_error("Failed to initialize device!");
//a##################################################
	dim3 dimg, dimb;
	dimg.x=(nz+2*npml+2*mm+BlockSize1-1)/BlockSize1;
	dimg.y=(nx+2*npml+2*mm+BlockSize2-1)/BlockSize2;
	dimb.x=BlockSize1;
	dimb.y=BlockSize2;
//a##################################################
	hipMalloc(&s_wlt, nt*sizeof(float));
	hipMalloc(&c, mm*sizeof(float));
	hipMalloc(&s_vv, nnz*nnx*sizeof(float));
	hipMalloc(&s_vx, nnz*nnx*sizeof(float));
	hipMalloc(&s_vn, nnz*nnx*sizeof(float));
	hipMalloc(&mut_vv, nnz*nnx*sizeof(float));
	hipMalloc(&mut_vx, nnz*nnx*sizeof(float));
	hipMalloc(&mut_vn, nnz*nnx*sizeof(float));
	hipMalloc(&s_p0, nnz*nnx*sizeof(float));
	hipMalloc(&s_p1, nnz*nnx*sizeof(float));
	hipMalloc(&s_q0, nnz*nnx*sizeof(float));
	hipMalloc(&s_q1, nnz*nnx*sizeof(float));
	hipMalloc(&mut_p0, nnz*nnx*sizeof(float));
	hipMalloc(&mut_p1, nnz*nnx*sizeof(float));
	hipMalloc(&mut_q0, nnz*nnx*sizeof(float));
	hipMalloc(&mut_q1, nnz*nnx*sizeof(float));
	hipMalloc(&g_p0, nnz*nnx*sizeof(float));
	hipMalloc(&g_p1, nnz*nnx*sizeof(float));
	hipMalloc(&g_q0, nnz*nnx*sizeof(float));
	hipMalloc(&g_q1, nnz*nnx*sizeof(float));
//a##################################################
	hipMalloc(&mig_is, nnz*nnx*sizeof(float));
	hipMalloc(&mig_ns, nnz*nnx*sizeof(float));
	hipMalloc(&illum_is, nnz*nnx*sizeof(float));
	hipMalloc(&illum_ns, nnz*nnx*sizeof(float));
//a##################################################
	hipMalloc(&adcigs_Dev, nz*na*nx*sizeof(float));
	hipMalloc(&angle_count_Dev, na*sizeof(int));
//a##################################################
	hipMalloc(&szx, ns*sizeof(int));
	hipMalloc(&gzx, ng*sizeof(int));
	hipMalloc(&coo_left , nz*sizeof(int));
	hipMalloc(&coo_right, nz*sizeof(int));
	hipMalloc(&coo_up   , nx*sizeof(int));
	hipMalloc(&coo_down , nx*sizeof(int));
//a##################################################
	hipMalloc(&cal_it_Dev, ng*sizeof(float));	
	hipMalloc(&cal_nt_Dev, ng*nt*sizeof(float));
	hipMalloc(&obs_it_Dev, ng*sizeof(float));	
	hipMalloc(&obs_nt_Dev, ng*nt*sizeof(float));
	hipMalloc(&mut_nt_Dev, ng*nt*sizeof(float));
	hipMalloc(&s_p_bndr, (2*nx+2*nz)*nt*sizeof(float)); 
	hipMalloc(&s_q_bndr, (2*nx+2*nz)*nt*sizeof(float)); 
//a##################################################
	cuda_ricker_wavelet<<<(nt+511)/512, 512>>>(s_wlt, fm, dt, nt);
	check_gpu_error("Failed to allocate memory for variables!");
//a##################################################
	hipMemcpy(s_vv, vv, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(s_vx, vx, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(s_vn, vn, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
//a##################################################
       cuda_mut_v<<<(nnx*nnz+511)/512, 512>>>(s_vv, s_vx, s_vn, mut_vv, mut_vx, mut_vn, nx, nz);
//a##################################################
       cuda_cal_c<<<1, 1>>>(c);
	cuda_set_s<<<1, ns>>>(szx, fsz, fsx, dsz, dsx, ns, nz, nx);
	cuda_set_g<<<(ng+511)/512,512>>>(gzx, ng, nz, nx);
       cuda_set_cooLR<<<(nz+511)/512,512>>>(coo_left, coo_right, nx, nz);
       cuda_set_cooUD<<<(nx+511)/512,512>>>(coo_up,   coo_down,  nx, nz);
//a##################################################
	clock_t t0, t1;
	t0 = clock();
//a##################################################
	hipMemset(adcigs_Dev, 0, nz*na*nx*sizeof(float));
	hipMemset(mig_ns, 0, nnz*nnx*sizeof(float));
	hipMemset(illum_ns, 0, nnz*nnx*sizeof(float));
	hipMemset(angle_count_Dev, 0, na*sizeof(int));
     for(is=0; is<ns; is++){
	  hipMemset(s_p0, 0, nnz*nnx*sizeof(float));
	  hipMemset(s_p1, 0, nnz*nnx*sizeof(float));
	  hipMemset(s_q0, 0, nnz*nnx*sizeof(float));
	  hipMemset(s_q1, 0, nnz*nnx*sizeof(float));
	  hipMemset(mut_p0, 0, nnz*nnx*sizeof(float));
	  hipMemset(mut_p1, 0, nnz*nnx*sizeof(float));
	  hipMemset(mut_q0, 0, nnz*nnx*sizeof(float));
	  hipMemset(mut_q1, 0, nnz*nnx*sizeof(float));
	  hipMemset(g_p0, 0, nnz*nnx*sizeof(float));
	  hipMemset(g_p1, 0, nnz*nnx*sizeof(float));
	  hipMemset(g_q0, 0, nnz*nnx*sizeof(float));
	  hipMemset(g_q1, 0, nnz*nnx*sizeof(float));
	  hipMemset(mig_is, 0, nnz*nnx*sizeof(float));
	  hipMemset(illum_is, 0, nnz*nnx*sizeof(float));
	  hipMemset(cal_it_Dev, 0, ng*sizeof(float));
	  hipMemset(cal_nt_Dev, 0, ng*nt*sizeof(float));
	  hipMemset(obs_it_Dev, 0, ng*sizeof(float));
	  hipMemset(obs_nt_Dev, 0, ng*nt*sizeof(float));
	  hipMemset(obs_it_Dev, 0, ng*sizeof(float));
	  hipMemset(mut_nt_Dev, 0, ng*nt*sizeof(float));
//a##################################################
         //cuda_iso_source_ring<<<(nnz*nnx+511)/512,512>>>(s_vv, s_vx, s_vn, nx, nz, &szx[is], 5, 15);
//a####################################################################################################
         printf("##  >>  is =%3d\n",is);
	  for(it=0; it<nt; it++){
	    cuda_add_source<<<1,1>>>(true, s_p1, &s_wlt[it], &szx[is], 1);
	    cuda_add_source<<<1,1>>>(true, s_q1, &s_wlt[it], &szx[is], 1);
	    cuda_step_fd2d<<<dimg,dimb>>>(s_p0, s_p1, s_q0, s_q1, s_vv, s_vx, s_vn, _dz2, _dx2, nz, nx,
                                        true, &szx[is], r, R);
           cuda_absorb_bndr<<<dimg,dimb>>>(s_p0, s_p1, s_q0, s_q1, nz, nx, -0.25);
           cuda_s_bndr<<<((2*nx+2*nz)+511)/512,512>>>(&s_p_bndr[it*(2*nx+2*nz)], &s_q_bndr[it*(2*nx+2*nz)], 
                                                      s_p0, s_q0, coo_left, coo_right, coo_up, coo_down, nz, nx, true);
	    ptr=s_p0; s_p0=s_p1; s_p1=ptr;
	    ptr=s_q0; s_q0=s_q1; s_q1=ptr;
//a##################################################
           cuda_cal_illum_matrix<<<(nnx*nnz+511)/512, 512>>>(illum_is, s_p0, nx, nz);
//a##################################################
	    cuda_record<<<(ng+511)/512, 512>>>(s_p0, cal_it_Dev, gzx, ng, true);
           cuda_trans_x2tx<<<(ng+511)/512, 512>>>(cal_it_Dev, cal_nt_Dev, it, nt, ng, true);
//a####################################################################################################
	    cuda_add_source<<<1,1>>>(true, mut_p1, &s_wlt[it], &szx[is], 1);
	    cuda_add_source<<<1,1>>>(true, mut_q1, &s_wlt[it], &szx[is], 1);
	    cuda_step_fd2d<<<dimg,dimb>>>(mut_p0, mut_p1, mut_q0, mut_q1, mut_vv, mut_vx, mut_vn, _dz2, _dx2, nz, nx,
                                        true, &szx[is], r, R);
           cuda_absorb_bndr<<<dimg,dimb>>>(mut_p0, mut_p1, mut_q0, mut_q1, nz, nx, -0.25);

	    ptr=mut_p0; mut_p0=mut_p1; mut_p1=ptr;
	    ptr=mut_q0; mut_q0=mut_q1; mut_q1=ptr;
//a##################################################
	    cuda_record<<<(ng+511)/512, 512>>>(mut_p0, cal_it_Dev, gzx, ng, true);
           cuda_trans_x2tx<<<(ng+511)/512, 512>>>(cal_it_Dev, mut_nt_Dev, it, nt, ng, true);
//a##################################################
	    if(is==0&&it!=0&&it%kt==0&&flag_snap){
	      hipMemcpy(vv, s_p0, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
	      window3d(temp, vv, nz, nx);
	      fwrite(temp, sizeof(float),nz*nx, fpsnap);	  
             }
            }
//a##################################################
         //cuda_mute_direct<<<(ng*nt+511)/512,512>>>(cal_nt_Dev, nx, nz, dx, dz, nt, dt, fm, vmute, &szx[is], 30);
//a##################################################
           cuda_difference<<<(ng*nt+511)/512, 512>>>(cal_nt_Dev, mut_nt_Dev, ng, nt);
//a##################################################
	  hipMemcpy(cal_nt_Hos, cal_nt_Dev, ng*nt*sizeof(float), hipMemcpyDeviceToHost);
         fseek(fpshotcal,is*ng*nt*sizeof(float),0);
	  fwrite(cal_nt_Hos, sizeof(float), ng*nt, fpshotcal);
//a##################################################
         for(i=0;i<ng*nt;i++)
               obs_nt_Hos[i]=cal_nt_Hos[i];
	  hipMemcpy(obs_nt_Dev, obs_nt_Hos, ng*nt*sizeof(float), hipMemcpyHostToDevice);
//a####################################################################################################
	  hipMemset(s_p0, 0, nnz*nnx*sizeof(float));
	  hipMemset(s_p1, 0, nnz*nnx*sizeof(float));
	  hipMemset(s_q0, 0, nnz*nnx*sizeof(float));
	  hipMemset(s_q1, 0, nnz*nnx*sizeof(float));
//a##################################################
	  for(it=nt-1; it>=0; it--){
//a##################################################
	    ptr=s_p0; s_p0=s_p1; s_p1=ptr;
	    ptr=s_q0; s_q0=s_q1; s_q1=ptr;
           cuda_s_bndr<<<((2*nx+2*nz)+511)/512,512>>>(&s_p_bndr[it*(2*nx+2*nz)], &s_q_bndr[it*(2*nx+2*nz)], 
                                                      s_p1, s_q1, coo_left, coo_right, coo_up, coo_down, nz, nx, false);
           cuda_step_fd2d<<<dimg,dimb>>>(s_p0, s_p1, s_q0, s_q1, s_vv, s_vx, s_vn, _dz2, _dx2, nz, nx,
                                        false, NULL, NULL, NULL);
           cuda_absorb_bndr<<<dimg,dimb>>>(s_p0, s_p1, s_q0, s_q1, nz, nx, -0.25);
//a##################################################
	  //  if(is==0&&it!=0&&it%kt==0&&flag_snap){
	  //    hipMemcpy(vv, s_p0, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
	  //    window3d(temp, vv, nz, nx);
	  //    fwrite(temp, sizeof(float),nz*nx, fpsnap);	  
          //   }
//a##################################################
           cuda_trans_x2tx<<<(ng+511)/512, 512>>>(obs_it_Dev, obs_nt_Dev, it, nt, ng, false);
	    cuda_record<<<(ng+511)/512, 512>>>(g_p1, obs_it_Dev, gzx, ng, false);
	    cuda_record<<<(ng+511)/512, 512>>>(g_q1, obs_it_Dev, gzx, ng, false);
           cuda_step_fd2d<<<dimg,dimb>>>(g_p0, g_p1, g_q0, g_q1, s_vv, s_vx, s_vn, _dz2, _dx2, nz, nx,
                                        false, NULL, NULL, NULL);
           cuda_absorb_bndr<<<dimg,dimb>>>(g_p0, g_p1, g_q0, g_q1, nz, nx, -0.25);
	    ptr=g_p0; g_p0=g_p1; g_p1=ptr;
	    ptr=g_q0; g_q0=g_q1; g_q1=ptr;
           cuda_cal_illum_matrix<<<(nnx*nnz+511)/512, 512>>>(illum_is, g_p0, nx, nz);
//a##################################################
	  //  if(is==0&&it!=0&&it%kt==0&&flag_snap){
	  //    hipMemcpy(vv, g_p0, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
	  //    window3d(temp, vv, nz, nx);
	  //    fwrite(temp, sizeof(float),nz*nx, fpsnap);	  
          //   }
//a##################################################
           cuda_cal_corr<<<(nnx*nnz+511)/512, 512>>>(mig_is, s_p1, g_p1, nx, nz);
//a##################################################
           cuda_poynting_adcigs<<<(nnx*nnz+511)/512, 512>>>(adcigs_Dev, s_p0, s_q0, g_p0, g_q0, nx, nz, na, _dx2, _dz2, 
                                                            illum_is, angle_count_Dev, c);
          }
//a##################################################
          hipMemcpy(vv, illum_is, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
	   window3d(temp, vv, nz, nx);
          fseek(fpillumis,is*nx*nz*sizeof(float),0);
	   fwrite(temp, sizeof(float),nz*nx, fpillumis);
//a##################################################
          cuda_illumination<<<(nnx*nnz+511)/512, 512>>>(mig_ns, mig_is, illum_ns, illum_is, nx, nz);
//a##################################################
          hipMemcpy(vv, mig_is, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
	   window3d(temp, vv, nz, nx);
          fseek(fpmigis,is*nx*nz*sizeof(float),0);
	   fwrite(temp, sizeof(float),nz*nx, fpmigis);
      }//end of IS loop
//a##################################################
     hipMemcpy(vv, mig_ns, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
     window3d(temp, vv, nz, nx);
     if(flag_laplace){
          laplace_filter(1, nz, nx, temp, v0);
          fwrite(v0, sizeof(float),nz*nx, fpmigns);
     }else{ fwrite(temp, sizeof(float),nz*nx, fpmigns); }
//a##################################################
     hipMemcpy(vv, illum_ns, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
     window3d(temp, vv, nz, nx);
     fwrite(temp, sizeof(float),nz*nx, fpillumns);
//a##################################################
     if(flag_adcigs_smooth)cuda_smooth_adcigs<<<(nx*nz+511)/512, 512>>>(adcigs_Dev, nx, nz, na, 10);
     hipMemcpy(adcigs_Hos, adcigs_Dev, nz*na*nx*sizeof(float), hipMemcpyDeviceToHost);
     fwrite(adcigs_Hos, sizeof(float),nz*na*nx, fpadcigs);
//a##################################################
     hipMemcpy(angle_count_Hos, angle_count_Dev, na*sizeof(int), hipMemcpyDeviceToHost);
     for(i=0;i<na;i++) printf("The number of %2d degree is %d.\n",i+1,angle_count_Hos[i]);
//a##################################################


     t1 = clock();
     printf("total %d shots: %f (s)\n", ns, ((float)(t1-t0))/CLOCKS_PER_SEC);

	/* free memory on device */
	hipFree(c);
	hipFree(s_wlt);
	hipFree(s_vv);
	hipFree(s_vx);
	hipFree(s_vn);
	hipFree(s_p0);
	hipFree(s_p1);
	hipFree(s_q0);
	hipFree(s_q1);
	hipFree(mut_vv);
	hipFree(mut_vx);
	hipFree(mut_vn);
	hipFree(mut_p0);
	hipFree(mut_p1);
	hipFree(mut_q0);
	hipFree(mut_q1);
	hipFree(g_p0);
	hipFree(g_p1);
	hipFree(g_q0);
	hipFree(g_q1);
	hipFree(mig_is);
	hipFree(mig_ns);
	hipFree(illum_is);
	hipFree(illum_ns);
	hipFree(szx);
	hipFree(gzx);
	hipFree(cal_it_Dev);
	hipFree(cal_nt_Dev);
	hipFree(obs_it_Dev);
	hipFree(obs_nt_Dev);
	hipFree(mut_nt_Dev);
	hipFree(adcigs_Dev);
	hipFree(angle_count_Dev);

	free(v0);
	free(temp);
	free(vv);
	free(vx);
	free(vn);
	free(cal_nt_Hos);
	free(obs_nt_Hos);
	free(adcigs_Hos);
	free(angle_count_Hos);

       fclose(fpvel);
       if(flag_snap) fclose(fpsnap);
       fclose(fpshotcal);
       fclose(fpshotobs);
       fclose(fpmigis);
       fclose(fpmigns);
       fclose(fpillumis);
       fclose(fpillumns);
       fclose(fpadcigs);

    	exit (0);
}

