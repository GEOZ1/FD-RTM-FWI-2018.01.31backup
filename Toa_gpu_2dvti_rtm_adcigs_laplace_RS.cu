#include "hip/hip_runtime.h"
//a#########################################################
//a##         2D Acoustic VTI Medium RTM   
//a##  Ps : P + sv wave and get rid of sv        
//a##       GPU(CUDA) ,poynting adcigs, read shot
//a##
//a##/*a***************************
//a##Function for VTI medium modeling,2017.2.13
//a##
//a## Ps:  the function of modeling following:
//a##      
//a##          du/dt=1/rho*dp/dx , 
//a##          dw/dt=1/rho*dq/dz ,  
//a##          dp/dt=rho*vpx^2*du/dx+rho*vp*vpn*dw/dz ,
//a##          dq/dt=rho*vp*vpn*du/dx+rho*vp^2*dw/dz ,
//a##                     vpx^2=vp^2*(1+2*epsilon);
//a##                     vpn^2=vp^2*(1+2*delta);
//a##*********a*******************/
//a##                        first code: 2017.2.15
//a##                    adcigs process: 2017.4.13
//a##
//a##                                  code by Rong Tao 
//a##                                     
//a#########################################################
#include<stdio.h>
#include<malloc.h>
#include<math.h>
#include<stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define pi 3.141592653

#define mm 4

//__constant__ float c[mm]={1.125,-0.04166667};/*mm==2*/
//__constant__ float c[mm]={1.1718750,-0.065104167,0.0046875};/*mm==3*/
__constant__ float c[mm]={1.196289,-0.0797526,0.009570313,-0.0006975447};/*mm==4*/
//__constant__ float c[mm]={1.211243,-0.08972168,0.01384277,-0.00176566,0.0001186795};/*mm==5*/


__device__ float d0;
//a################################################################################
void check_gpu_error (const char *msg) 
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) { 
	printf("Cuda error: %s: %s\n", msg, hipGetErrorString (err)); 
	exit(0);   
    }
}
/*************func**************/
void laplace_filter(int adj, int nz, int nx, float *in, float *out)
/*< linear operator, come from Madagascar Mlaplac2>*/
{
    int iz,ix,j;
    for (j=0;j<nx*nz;j++) out[j]=0.0;
    for (ix=0; ix < nx; ix++) {
	for (iz=0; iz < nz; iz++) {
	    j = iz+ix*nz;
	    if (iz > 0) {
		if (adj) {
		    out[j-1] -= in[j];
		    out[j]   += in[j];
		} else {
		    out[j] += in[j] - in[j-1];
		}
	    }
	    if (iz < nz-1) {
		if (adj) {
		    out[j+1] -= in[j];
		    out[j]   += in[j];
		} else {
		    out[j] += in[j] - in[j+1];
		}
	    }

	    if (ix > 0) {
		if (adj) {
		    out[j-nz] -= in[j];
		    out[j]    += in[j];
		} else {
		    out[j] += in[j] - in[j-nz];
		}
	    }
	    if (ix < nx-1) {
		if (adj) {
		    out[j+nz] -= in[j];
		    out[j]    += in[j];
		} else {
		    out[j] += in[j] - in[j+nz];
		}
	    }
	}
    }
}
/*************func**************/
__global__ void add_source(float pfac,float xsn,float zsn,int nx,int nz,int nnx,int nnz,float dt,float t,
                        float favg,int wtype,int npml,int is,int ds,float *P,float *Q)
/*< generate ricker wavelet with time deley >*/
{
       int ixs,izs;
       float x_,xx_,tdelay,ts,source=0.0,fs; 
  
       tdelay=1.0/favg;
       ts=t-tdelay;
       fs=xsn+(is-1)*ds;

	if(wtype==1)//ricker wavelet
	{
          x_=favg*ts;
          xx_=x_*x_;
          source=(1-2*pi*pi*(xx_))*exp(-(pi*pi*xx_));
	}else if(wtype==2){//derivative of gaussian
          x_=(-4)*favg*favg*pi*pi/log(0.1);
          source=(-2)*pi*pi*ts*exp(-x_*ts*ts);
        }else if(wtype==3){//derivative of gaussian
          x_=(-1)*favg*favg*pi*pi/log(0.1);
          source=exp(-x_*ts*ts);
        }

       if(t<=2*tdelay)
       {         
	     ixs = (int)(fs+0.5)+npml-1;
            izs = (int)(zsn+0.5)+npml-1;
            P[ixs*nnz+izs]+=pfac*source;
            Q[ixs*nnz+izs]+=pfac*source;
       }
}
/*******************func*********************/
__global__ void update_vel(int nx,int nz,int nnx,int nnz,int npml,float dt,float dx,float dz,
                           float *u0,float *w0,float *u1,float *w1,float *P,float *Q,
                           float *coffx1,float *coffx2,float *coffz1,float *coffz2)
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;

	int ix,iz,im;
	float dtx,dtz,xx,zz;

        ix=id/nnz;
        iz=id%nnz;

		 dtx=dt/dx;
		 dtz=dt/dz;
               if(id>=mm&&id<nnx*nnz-mm)
                 {
                   if(ix>=mm&&ix<(nnx-mm)&&iz>=mm&&iz<(nnz-mm))
                    {
                     xx=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                      {
                        xx+=c[im]*(P[id+(im+1)*nnz]-P[id-im*nnz]);
                        zz+=c[im]*(Q[id+im+1]      -Q[id-im]);
                      }
                     u1[id]=coffx2[ix]*u0[id]-coffx1[ix]*dtx*xx;
                     w1[id]=coffz2[iz]*w0[id]-coffz1[iz]*dtz*zz;
                   }
                 }
}
/*******************func***********************/
__global__ void update_stress(int nx,int nz,int nnx,int nnz,float dt,float dx,float dz,
                           float *u1,float *w1,float *P,float *Q,float *vp,int npml,
                           float *px1,float *px0,float *pz1,float *pz0,float *qx1,float *qx0,float *qz1,float *qz0,
                           float *acoffx1,float *acoffx2,float *acoffz1,float *acoffz2,
                           float *delta,float *epsilon,int fs,int ds,int zs,int is,bool SV)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

	int im,ix,iz,rx,rz,R=15,r=5;
	float dtx,dtz, xx,zz,ee,dd;

        ix=id/nnz;
        iz=id%nnz;

               dtx=dt/dx;
		 dtz=dt/dz;
               if(id>=mm&&id<nnx*nnz-mm)
                 {
/************************i****************************************/
/************************iso circle start*************************/
                   rx=ix-(fs+(is-1)*ds+npml);
                   rz=iz-(zs+npml);
                   if(SV){
                       if((rx*rx+rz*rz)<=R*R){
                           if((rx*rx+rz*rz)<=r*r){
                               ee = 0.0;
                               dd = 0.0;
                           }else{
                               ee = 0.5*(1-cos(pi*((sqrtf(rx*rx+rz*rz)-r)*4.0/(R*3.0-1))))*epsilon[id];
                               dd = 0.5*(1-cos(pi*((sqrtf(rx*rx+rz*rz)-r)*4.0/(R*3.0-1))))*delta[id]; 
                              }
                       }else{
                          ee=epsilon[id];
                          dd=delta[id];
                          }
                   }else{
                      ee=epsilon[id];
                      dd=delta[id];
                     }
/************************ iso circle end *************************/
/************************i****************************************/
                   if(ix>=mm&&ix<(nnx-mm)&&iz>=mm&&iz<(nnz-mm))
                     {
                     xx=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                       {
                        xx+=c[im]*(u1[id+im*nnz]-u1[id-(im+1)*nnz]);
                        zz+=c[im]*(w1[id+im]    -w1[id-im-1]);
                       }
                     px1[id]=acoffx2[ix]*px0[id]-acoffx1[ix]*vp[id]*vp[id]*(1+2*ee)*dtx*xx;
                     pz1[id]=acoffz2[iz]*pz0[id]-acoffz1[iz]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtz*zz;
                     qx1[id]=acoffx2[ix]*qx0[id]-acoffx1[ix]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtx*xx;
                     qz1[id]=acoffz2[iz]*qz0[id]-acoffz1[iz]*vp[id]*vp[id]*dtz*zz;

                     P[id]=px1[id]+pz1[id];
                     Q[id]=qx1[id]+qz1[id];
                   }
                 }
}                      
/********************func**********************/
__global__ void get_d0(float dx,float dz,int nnx,int nnz,int npml,float *vp)
{
   d0=10.0*vp[nnx*nnz/2]*log(100000.0)/(2.0*npml*((dx+dz)/2.0));
}
/*************func*******************/
void pad_vv(int nx,int nz,int nnx,int nnz,int npml,float *ee)
{
     int ix,iz,id;
 
    for(id=0;id<nnx*nnz;id++)
     {
       ix=id/nnz;
       iz=id%nnz;
       if(ix<npml){
           ee[id]=ee[npml*nnz+iz];  //left
       }else if(ix>=nnx-npml){
           ee[id]=ee[(nnx-npml-1)*nnz+iz];//right
       }
     }
    for(id=0;id<nnx*nnz;id++)
     {
       ix=id/nnz;
       iz=id%nnz;
       if(iz<npml){
           ee[id]=ee[ix*nnz+npml];//up
       }else if(iz>=nnz-npml){
           ee[id]=ee[ix*nnz+nnz-npml-1];//down
       }
     }
}

/*************func*******************/
__global__ void initial_coffe(float dt,int nn,float *coff1,float *coff2,float *acoff1,float *acoff2,int npml)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

           if(id<nn+2*npml)
            {
		 if(id<npml)
		 {   
			 coff1[id]=1.0/(1.0+(dt*d0*pow((npml-0.5-id)/npml,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt*d0*pow((npml-0.5-id)/npml,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);

		 }else if(id>=npml&&id<npml+nn){

			 coff1[id]=1.0;
			 coff2[id]=1.0;

			 acoff1[id]=1.0;
			 acoff2[id]=1.0;

		 }else{

			 coff1[id]=1.0/(1.0+(dt*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
		 }	
            }       
}
/*************func*******************/
__global__ void shot_record(int nnx, int nnz, int nx, int nz, int npml, int it, int nt, float *P, float *shot, bool flag)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

           if(id<nx)
            {
             if(flag){
               shot[it+nt*id]=P[npml+nnz*(id+npml)];
             }else{
               P[npml+nnz*(id+npml)]=shot[it+nt*id];
              }
            }       
}
/*************func*******************/
__global__ void wavefield_bndr(int nnx, int nnz, int nx, int nz, int npml, int it, int nt, 
                               float *P, float *Q, float *P_bndr, float *Q_bndr, bool flag)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

           if(id<2*nx+2*nz)
            {
            if(flag)/////////////////////////////////save boundary
             {
              if(id<nx){//up

               P_bndr[it*(2*nx+2*nz)+id]=P[npml-1+nnz*(id+npml)];
               Q_bndr[it*(2*nx+2*nz)+id]=Q[npml-1+nnz*(id+npml)];

              }else if(id>=nx&&id<(2*nx)){//down
   
               P_bndr[it*(2*nx+2*nz)+id]=P[npml+nz+1+nnz*(id-nx+npml)];
               Q_bndr[it*(2*nx+2*nz)+id]=Q[npml+nz+1+nnz*(id-nx+npml)];


              }else if(id>=(2*nx)&&id<(2*nx+nz)){//left

               P_bndr[it*(2*nx+2*nz)+id]=P[id-2*nx+npml+nnz*(npml-1)];
               Q_bndr[it*(2*nx+2*nz)+id]=Q[id-2*nx+npml+nnz*(npml-1)];

              }else if(id>=(2*nx+nz)){//right

               P_bndr[it*(2*nx+2*nz)+id]=P[id-2*nx-nz+npml+nnz*(npml+nx+1)];
               Q_bndr[it*(2*nx+2*nz)+id]=Q[id-2*nx-nz+npml+nnz*(npml+nx+1)];

                }
            }else{/////////////////////////////add boundary
              if(id<nx){//up

               P[npml-1+nnz*(id+npml)]=P_bndr[it*(2*nx+2*nz)+id];
               Q[npml-1+nnz*(id+npml)]=Q_bndr[it*(2*nx+2*nz)+id];

              }else if(id>=nx&&id<(2*nx)){//down
   
               P[npml+nz+1+nnz*(id-nx+npml)]=P_bndr[it*(2*nx+2*nz)+id];
               Q[npml+nz+1+nnz*(id-nx+npml)]=Q_bndr[it*(2*nx+2*nz)+id];


              }else if(id>=(2*nx)&&id<(2*nx+nz)){//left

               P[id-2*nx+npml+nnz*(npml-1)]=P_bndr[it*(2*nx+2*nz)+id];
               Q[id-2*nx+npml+nnz*(npml-1)]=Q_bndr[it*(2*nx+2*nz)+id];

              }else if(id>=(2*nx+nz)){//right

               P[id-2*nx-nz+npml+nnz*(npml+nx+1)]=P_bndr[it*(2*nx+2*nz)+id];
               Q[id-2*nx-nz+npml+nnz*(npml+nx+1)]=Q_bndr[it*(2*nx+2*nz)+id];

                }
             }
            }       
}
/*************func**************/    
__global__ void mute_directwave(int nx,int nt,float dt,float favg,
                     float dx,float dz,int fs,int ds,int zs,int is,
                     float *vp,float *epsilon,float *shot,int tt)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

    int mu_t,mu_nt;
    float mu_x,mu_z,mu_t0;

    int ix=id/nt;
    int it=id%nt;

   if(id<nx*nt)
   {
        mu_x=dx*abs(ix-fs-(is-1)*ds);
        mu_z=dz*zs;
        mu_t0=sqrtf(pow(mu_x,2)+pow(mu_z,2))/(vp[1]*sqrtf(1+2*epsilon[1]));
        mu_t=(int)(2.0/(dt*favg));
        mu_nt=(int)(mu_t0/dt)+mu_t+tt;

           if((it>(int)(mu_t0/dt)-tt)&&(it<mu_nt))
              shot[id]=0.0;
   }
}
/*************func**************/    
__global__ void cal_illumination(int nnx, int nnz, int nz, int npml, float *illumination, float *P, float *Q)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    int ix=id/nz;
    int iz=id%nz;

   if(id<nnx*nnz)
   {
      illumination[id]+=P[iz+npml+nnz*(ix+npml)]*P[iz+npml+nnz*(ix+npml)]
                         +Q[iz+npml+nnz*(ix+npml)]*Q[iz+npml+nnz*(ix+npml)];
      if(illumination[id]==0)illumination[id]=1.0;
   }
}
/*************func**************/    
__global__ void cal_migration(int nnx, int nnz, int nz, int npml, float *migration, float *s, float *g)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    int ix=id/nz;
    int iz=id%nz;

   if(id<nnx*nnz)
   {
      migration[id]+=s[iz+npml+nnz*(ix+npml)]*g[iz+npml+nnz*(ix+npml)];
   }
}
/*************func**************/    
__global__ void migration_illum(int nx, int nz, int npml, float *migration, float *illumination)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

   if(id<nx*nz)
   {
      migration[id]/=illumination[id];//*illumination[id];
   }
}
/*************func**************/    
__global__ void Poynting_Adcigs(int nnz, int nx, int nz, int npml, int na, int da, float *adcigs, 
                           float *s_P, float *s_Q, float *s_u, float *s_w, 
                           float *g_P, float *g_Q, float *g_u, float *g_w)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    int ix=id/nz;
    int iz=id%nz;

    int ia=0;

    float Ssx=-s_P[iz+npml+nnz*(ix+npml)]*s_u[iz+npml+nnz*(ix+npml)];
    float Ssz=-s_Q[iz+npml+nnz*(ix+npml)]*s_w[iz+npml+nnz*(ix+npml)];
    float Sgx= g_P[iz+npml+nnz*(ix+npml)]*g_u[iz+npml+nnz*(ix+npml)];
    float Sgz= g_Q[iz+npml+nnz*(ix+npml)]*g_w[iz+npml+nnz*(ix+npml)];

    float b1= Ssx*Ssx + Ssz*Ssz;
    float b2= Sgx*Sgx + Sgz*Sgz;
    float  a=(Ssx*Sgx + Ssz*Sgz)/(sqrtf(b1*b2)*(1 - 0.1));

   if(id<nx*nz)
   {
     if(a>=-1&&a<=1)
      {
         a=0.5*acosf(a)*180.0/pi;
         ia=(int)(a/(da*1.0));
         if(ia<na)
          {
             adcigs[iz+nz*ia+nz*na*(id/nz)] += s_P[iz+npml+nnz*(ix+npml)]*g_P[iz+npml+nnz*(ix+npml)]
                                                *cosf(ia*pi/180.0)*cosf(ia*pi/180.0)*cosf(ia*pi/180.0);
          }
      }
   }
}
/*************func**************/    
__global__ void adcigs_illum(int nx, int nz, int na, int da, float *adcigs, float *illumination)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    int ix=id/(nz*na);
    int iz=id%nz;

   if(id<nx*nz*na)
   {
      adcigs[id]/=illumination[iz+nz*ix];//*illumination[iz+nz*ix];
   }
}
/*************func**************/ 
void stk_adcigs(int nx,int nz,int na,float *adcigs,float *migration)
{
   int ix,iz,ia,id,ido;
   float stk;
   float *temp;

   temp=(float*)malloc(nz*nx*sizeof(float));

   for (ix=0; ix<nx; ix++)  {
       for (iz=0; iz<nz; iz++)  {
           stk=0.0;
           for (ia=0; ia<na; ia++)  {
                id=ix*na*nz+ia*nz+iz;
                stk+=adcigs[id];
             }
           ido=ix*nz+iz;
           temp[ido]=stk;
        }
   }
   laplace_filter(1,nz,nx,temp,migration);
}
/*************func**************/ 
void adcigs_smiled(int nx,int nz,int na,int dcdp,float *adcigs)   
{
   int ix,iz,ia,id,ido;
   float *temp;

   temp=(float*)malloc(nz*nx/dcdp*na*sizeof(float));
   for (ix=0; ix<nx; ix++)  {
       for (ia=0; ia<na; ia++)  {
           for (iz=0; iz<nz; iz++)  {
                id=ix*na*nz+ia*nz+iz;
                if(ix%dcdp==0) {
                      ido=ix/dcdp*na*nz+ia*nz+iz;
                      temp[ido]=adcigs[id];
                      adcigs[ido]=temp[ido];
                  }
             }
        }
   }

}
/*************func*******************/
void read_file(char FN1[],char FN2[],char FN3[],int nx,int nz,int nnx,int nnz,float dx,float dz,float favg,float dt,
               float *v,float *e,float *d,int npml)
{
		 int i,j,id;
               float vmax,  vmin,emax, emin, dmax, dmin,    H_min, dt_max, dxz_max, C, tmp;

		
		 FILE *fp1,*fp2,*fp3;
		 if((fp1=fopen(FN1,"rb"))==NULL){printf("error open <%s>!\n",FN1);exit(0);}
		 if((fp2=fopen(FN2,"rb"))==NULL){printf("error open <%s>!\n",FN2);exit(0);}
		 if((fp3=fopen(FN3,"rb"))==NULL){printf("error open <%s>!\n",FN3);exit(0);}

               vmin= 999999.9;
               vmax=-999999.9;
		 for(i=npml;i<nx+npml;i++)
		 {
			 for(j=npml;j<nz+npml;j++)
			 {
                            id=i*nnz+j;
				 fread(&v[id],4L,1,fp1);/* test the paras *///v[id]*=1.00;
				 fread(&e[id],4L,1,fp2);/* test the paras *///e[id]*=1.00;
				 fread(&d[id],4L,1,fp3);/* test the paras *///d[id]*=0.00;

                            if(vmax<v[id]) vmax = v[id];
			       if(vmin>v[id]) vmin = v[id];
                            if(emax<e[id]) emax = e[id];
			       if(emin>e[id]) emin = e[id];
                            if(dmax<d[id]) dmax = d[id];
			       if(dmin>d[id]) dmin = d[id];
			 }
		 }
		 fclose(fp1);
		 fclose(fp2);
		 fclose(fp3);
        printf("------------------------------------\n---\n");
        printf("---   Vmax=%.2f, Vmin=%.2f\n",vmax,vmin);
        printf("---   Emax=%.4f, Emin=%.4f\n",emax,emin);
        printf("---   Dmax=%.4f, Dmin=%.4f\n---\n",dmax,dmin);
        /*********boundary*********/
        pad_vv(nx,nz,nnx,nnz,npml,e);
        pad_vv(nx,nz,nnx,nnz,npml,d);
        pad_vv(nx,nz,nnx,nnz,npml,v); 
      
       H_min=dx<dz?dx:dz;
       dt_max = 0.5*H_min/vmin;
       dxz_max = vmax/favg*0.2;

       if(dxz_max<dz||dxz_max<dx){printf("---   You need have to redefine DX and DZ ! \n");exit(0);}
	if(dt_max<dt){printf("---   You need have to redefine DT ! \n");exit(0);}
       if (   favg >= vmin/( 5.0*(dx>dz?dx:dz) )   ||   favg >= vmin/( 5.0*(dx>dz?dx:dz) )  )
	             {printf("---   Non-dispersion relation not satisfied! \n");exit(0);}

	else if ( mm == 2 )     C = 0.857;
	else if ( mm == 3 )     C = 0.8;
	else if ( mm == 4 )     C = 0.777;
	else if ( mm == 5 )     C = 0.759;

       tmp = dt*vmax*sqrtf( 1.0/(dx*dx)+1.0/(dz*dz) );
       if ( tmp >= C){ printf("---   Stability condition not satisfied! tmp = %f, C = %f\n",tmp,C);exit(0);}
}
//a########################################################################
//a##                          Main Function                             ##
//a########################################################################
int main(int argc,char *argv[])
{
	int is, it, nx, nz, nnx, nnz, nt, wtype, na, da, dcdp;
	int ns, ds, fs, zs, npml;
	float dx, dz, dt, t, pfac, favg;
       float *coffx1,*coffx2,*coffz1,*coffz2,*acoffx1,*acoffx2,*acoffz1,*acoffz2;
	float *v, *e, *d;
	float *vp, *epsilon, *delta;
	float *s_u0, *s_u1, *s_px0, *s_qx0, *s_px1, *s_qx1;
       float *s_w0, *s_w1, *s_pz0, *s_qz0, *s_pz1, *s_qz1;
	float *g_u0, *g_u1, *g_px0, *g_qx0, *g_px1, *g_qx1;
       float *g_w0, *g_w1, *g_pz0, *g_qz0, *g_pz1, *g_qz1;
	float *s_P, *s_Q, *g_P, *g_Q, *shot_Dev, *shot_Hos, *P_bndr, *Q_bndr;
       float *migration, *illumination, *adcigs;
       float *Atemp;
       bool read;


       clock_t start, end;
/*************wavelet\boundary**************/
          wtype=1;npml=20;
/********** dat document ***********/

          char FN1[250]={"waxian_3layer/waxian_vel_1001_301.dat"};
          char FN2[250]={"waxian_3layer/waxian_epsilon_1001_301.dat"};
          char FN3[250]={"waxian_3layer/waxian_delta_1001_301.dat"};
	   char FN4[250]={"waxian_3layer/waxian_shot_obs.dat"};//shot obs
	   char FN5[250]={"waxian_3layer_2/waxian_v1.00e1.00d0.00_shot_cal.dat"};//shot cal
	   char FN6[250]={"waxian_3layer_2/waxian_v1.00e1.00d0.00_snap.dat"};//snap
	   char FN7[250]={"waxian_3layer_2/waxian_v1.00e1.00d0.00_migration.dat"};
	   char FN8[250]={"waxian_3layer_2/waxian_v1.00e1.00d0.00_illumination.dat"};
	   char FN9[250]={"waxian_3layer_2/waxian_v1.00e1.00d0.00_adcigs.dat"}; 
	  char FN10[250]={"waxian_3layer_2/waxian_v1.00e1.00d0.00_stkadcigs.dat"};
	  char FN11[250]={"waxian_3layer_2/waxian_v1.00e1.00d0.00_smiled_adcigs.dat"};

/********* parameters *************/
          read=true;/* true: read shot; flase: use right shot record */
/********* parameters *************/
          nx=1001;              
	   nz=301;         favg=60;     pfac=1000.0;

 	   dx=5.0;   
          dz=5.0;   
     
	   nt=2501;    
          dt=0.0005;
     
          ns=500;       
          fs=nx/ns/2;      
          ds=nx/ns;
          zs=1;   

          na=70; 
          da=1;  
          dcdp=25;

     /*     char FN1[250]={"sshengli_duankuai/sshengli_vel_1000_550.dat"};      
          char FN2[250]={"sshengli_duankuai/sshengli_epsilon_1000_550.dat"};
          char FN3[250]={"sshengli_duankuai/sshengli_delta_1000_550.dat"}; 
	   char FN4[250]={"sshengli_duankuai/sshengli_shot_obs.dat"};//shot obs
	   char FN5[250]={"sshengli_duankuai/sshengli_v1.00e0.60d0.60_shot_cal.dat"};//shot cal
	   char FN6[250]={"sshengli_duankuai/sshengli_v1.00e0.60d0.60_snap.dat"};//snap
	   char FN7[250]={"sshengli_duankuai/sshengli_v1.00e0.60d0.60_migration.dat"};
	   char FN8[250]={"sshengli_duankuai/sshengli_v1.00e0.60d0.60_illumination.dat"};
	   char FN9[250]={"sshengli_duankuai/sshengli_v1.00e0.60d0.60_adcigs.dat"}; 
	  char FN10[250]={"sshengli_duankuai/sshengli_v1.00e0.60d0.60_stkadcigs.dat"};
	  char FN11[250]={"sshengli_duankuai/sshengli_v1.00e0.60d0.60_smiled_adcigs.dat"};*/

/********* parameters *************/
        //  read=true;/* true: read shot; flase: use right shot record */
/********* parameters *************/
     /*     nx=1000;              
	   nz=550;         favg=40;     pfac=1000.0;

 	   dx=5.0;   
          dz=5.0;   
     
	   nt=4501;    
          dt=0.0005;
     
          ns=500;       
          fs=nx/ns/2;      
          ds=nx/ns;
          zs=1;   

          na=70; 
          da=1;  
          dcdp=25;  */

/********aaa************/  
	 FILE *fpsnap, *fpobs, *fpcal, *fpmig, *fpillum, *fpadcigs, *fpadcigs2, *fpstk;
        if((fpobs=fopen(FN4,"rb"))==NULL){printf("error open <%s>!\n",FN4);exit(0);}
        fpcal=fopen(FN5,"wb");
        fpsnap=fopen(FN6,"wb");
        fpmig=fopen(FN7,"wb");
        fpillum=fopen(FN8,"wb");
        fpadcigs=fopen(FN9,"wb");
        fpstk=fopen(FN10,"wb");
        fpadcigs2=fopen(FN11,"wb");
/*************v***************/ 
          nnx=nx+2*npml;
          nnz=nz+2*npml;
/************a*************/
    	 Atemp=(float*)malloc(nz*nx*na*sizeof(float));

    	 v=(float*)malloc(nnz*nnx*sizeof(float));
    	 e=(float*)malloc(nnz*nnx*sizeof(float));
    	 d=(float*)malloc(nnz*nnx*sizeof(float));
    	 shot_Hos=(float*)malloc(nt*nx*sizeof(float));
        read_file(FN1,FN2,FN3,nx,nz,nnx,nnz,dx,dz,favg,dt,v,e,d,npml);
/****************************/

        hipSetDevice(0);// initialize device, default device=0;
	 check_gpu_error("Failed to initialize device!");

/****************************/
        hipMalloc(&vp, nnz*nnx*sizeof(float));
        hipMalloc(&epsilon, nnz*nnx*sizeof(float));
        hipMalloc(&delta, nnz*nnx*sizeof(float));
	 hipMemcpy(vp, v, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(epsilon, e, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(delta, d, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
/****************************/
        hipMalloc(&s_u0, nnz*nnx*sizeof(float));    hipMalloc(&s_u1, nnz*nnx*sizeof(float));
        hipMalloc(&s_w0, nnz*nnx*sizeof(float));    hipMalloc(&s_w1, nnz*nnx*sizeof(float));

        hipMalloc(&s_P, nnz*nnx*sizeof(float));     hipMalloc(&s_Q, nnz*nnx*sizeof(float));

        hipMalloc(&s_px0, nnz*nnx*sizeof(float));   hipMalloc(&s_px1, nnz*nnx*sizeof(float));
        hipMalloc(&s_pz0, nnz*nnx*sizeof(float));   hipMalloc(&s_pz1, nnz*nnx*sizeof(float));
        hipMalloc(&s_qx0, nnz*nnx*sizeof(float));   hipMalloc(&s_qx1, nnz*nnx*sizeof(float));
        hipMalloc(&s_qz0, nnz*nnx*sizeof(float));   hipMalloc(&s_qz1, nnz*nnx*sizeof(float));

        hipMalloc(&g_u0, nnz*nnx*sizeof(float));    hipMalloc(&g_u1, nnz*nnx*sizeof(float));
        hipMalloc(&g_w0, nnz*nnx*sizeof(float));    hipMalloc(&g_w1, nnz*nnx*sizeof(float));

        hipMalloc(&g_P, nnz*nnx*sizeof(float));     hipMalloc(&g_Q, nnz*nnx*sizeof(float));

        hipMalloc(&g_px0, nnz*nnx*sizeof(float));   hipMalloc(&g_px1, nnz*nnx*sizeof(float));
        hipMalloc(&g_pz0, nnz*nnx*sizeof(float));   hipMalloc(&g_pz1, nnz*nnx*sizeof(float));
        hipMalloc(&g_qx0, nnz*nnx*sizeof(float));   hipMalloc(&g_qx1, nnz*nnx*sizeof(float));
        hipMalloc(&g_qz0, nnz*nnx*sizeof(float));   hipMalloc(&g_qz1, nnz*nnx*sizeof(float));

        hipMalloc(&coffx1, nnx*sizeof(float));     hipMalloc(&coffx2, nnx*sizeof(float));
        hipMalloc(&coffz1, nnz*sizeof(float));     hipMalloc(&coffz2, nnz*sizeof(float));
        hipMalloc(&acoffx1, nnx*sizeof(float));    hipMalloc(&acoffx2, nnx*sizeof(float));
        hipMalloc(&acoffz1, nnz*sizeof(float));    hipMalloc(&acoffz2, nnz*sizeof(float));

        hipMalloc(&shot_Dev, nx*nt*sizeof(float));
        hipMalloc(&P_bndr, nt*(2*nx+2*nz)*sizeof(float));
        hipMalloc(&Q_bndr, nt*(2*nx+2*nz)*sizeof(float));

        hipMalloc(&migration, nz*nx*sizeof(float)); 
        hipMalloc(&illumination, nz*nx*sizeof(float));
        hipMalloc(&adcigs, nz*na*nx*sizeof(float));
/******************************/
	 check_gpu_error("Failed to allocate memory for variables!");

        get_d0<<<1, 1>>>(dx, dz, nnx, nnz, npml, vp);
        initial_coffe<<<(nnx+511)/512, 512>>>(dt,nx,coffx1,coffx2,acoffx1,acoffx2,npml);
        initial_coffe<<<(nnz+511)/512, 512>>>(dt,nz,coffz1,coffz2,acoffz1,acoffz2,npml);

        hipMemset(migration, 0, nz*nx*sizeof(float)); 
        hipMemset(illumination, 0, nz*nx*sizeof(float));
        hipMemset(adcigs, 0, nz*na*nx*sizeof(float));
        printf("--------------------------------------------------------\n");
        printf("---");   
        start = clock();                                  
/**********IS Loop start*******/
   for(is=1;is<=ns;is++)	
    {     
         printf("\n---   IS=%3d  ",is);

     hipMemset(s_u0, 0, nnz*nnx*sizeof(float));     hipMemset(s_u1, 0, nnz*nnx*sizeof(float));
     hipMemset(s_w0, 0, nnz*nnx*sizeof(float));     hipMemset(s_w1, 0, nnz*nnx*sizeof(float));

     hipMemset(s_P, 0, nnz*nnx*sizeof(float));      hipMemset(s_Q, 0, nnz*nnx*sizeof(float));

     hipMemset(s_px0, 0, nnz*nnx*sizeof(float));    hipMemset(s_px1, 0, nnz*nnx*sizeof(float));
     hipMemset(s_pz0, 0, nnz*nnx*sizeof(float));    hipMemset(s_pz1, 0, nnz*nnx*sizeof(float));
     hipMemset(s_qx0, 0, nnz*nnx*sizeof(float));    hipMemset(s_qx1, 0, nnz*nnx*sizeof(float));
     hipMemset(s_qz0, 0, nnz*nnx*sizeof(float));    hipMemset(s_qz1, 0, nnz*nnx*sizeof(float));

     hipMemset(g_u0, 0, nnz*nnx*sizeof(float));     hipMemset(g_u1, 0, nnz*nnx*sizeof(float));
     hipMemset(g_w0, 0, nnz*nnx*sizeof(float));     hipMemset(g_w1, 0, nnz*nnx*sizeof(float));

     hipMemset(g_P, 0, nnz*nnx*sizeof(float));      hipMemset(g_Q, 0, nnz*nnx*sizeof(float));

     hipMemset(g_px0, 0, nnz*nnx*sizeof(float));    hipMemset(g_px1, 0, nnz*nnx*sizeof(float));
     hipMemset(g_pz0, 0, nnz*nnx*sizeof(float));    hipMemset(g_pz1, 0, nnz*nnx*sizeof(float));
     hipMemset(g_qx0, 0, nnz*nnx*sizeof(float));    hipMemset(g_qx1, 0, nnz*nnx*sizeof(float));
     hipMemset(g_qz0, 0, nnz*nnx*sizeof(float));    hipMemset(g_qz1, 0, nnz*nnx*sizeof(float));

     hipMemset(shot_Dev, 0, nt*nx*sizeof(float));
     hipMemset(P_bndr, 0, nt*(2*nx+2*nz)*sizeof(float));
     hipMemset(Q_bndr, 0, nt*(2*nx+2*nz)*sizeof(float));

/*a***********************************Forward*******************************************/
     for(it=0,t=dt;it<nt;it++,t+=dt)
     { 
      //if(it==0)printf(" > F >",is,it);
       /*a#####################a*/
       /*a##     Forward     ##a*/
       /*a#####################a*/
	 add_source<<<1,1>>>(pfac,fs,zs,nx,nz,nnx,nnz,dt,t,favg,wtype,npml,is,ds,s_P,s_Q);
        update_vel<<<(nnx*nnz+511)/512, 512>>>(nx,nz,nnx,nnz,npml,dt,dx,dz,
                                               s_u0,s_w0,s_u1,s_w1,s_P,s_Q,coffx1,coffx2,coffz1,coffz2);
        update_stress<<<(nnx*nnz+511)/512, 512>>>(nx,nz,nnx,nnz,dt,dx,dz,s_u1,s_w1,s_P,s_Q,vp,npml,
                                                  s_px1,s_px0,s_pz1,s_pz0,s_qx1,s_qx0,s_qz1,s_qz0,
                                                  acoffx1,acoffx2,acoffz1,acoffz2,delta,epsilon,fs,ds,zs,is,true);
        s_u0=s_u1; s_w0=s_w1; s_px0=s_px1; s_pz0=s_pz1; s_qx0=s_qx1; s_qz0=s_qz1; 

        shot_record<<<(nx+511)/512, 512>>>(nnx, nnz, nx, nz, npml, it, nt, s_P, shot_Dev, true);
        wavefield_bndr<<<((2*nx+2*nz)+511)/512,512>>>(nnx, nnz, nx, nz, npml, it, nt, s_P, s_Q, P_bndr, Q_bndr, true);
        cal_illumination<<<(nx*nz+511)/512, 512>>>(nnx, nnz, nz, npml, illumination, s_P, s_Q);

     /*      if((is==1)&&(it%300==0))
            {
	       hipMemcpy(e, s_P, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
              fwrite(e,4L,nnx*nnz,fpsnap);
            }*/
     }//it loop end
      mute_directwave<<<(nx*nt+511)/512, 512>>>(nx,nt,dt,favg,dx,dz,fs,ds,zs,is,vp,epsilon,shot_Dev,20);
      hipMemcpy(shot_Hos, shot_Dev, nt*nx*sizeof(float), hipMemcpyDeviceToHost);
      //fseek(fpcal,(is-1)*nt*nx*sizeof(float),0);
      //fwrite(shot_Hos,sizeof(float),nt*nx,fpcal);

    if(read){
         fseek(fpobs,(is-1)*nt*nx*sizeof(float),0);
         fread(shot_Hos,sizeof(float),nt*nx,fpobs);
         hipMemcpy(shot_Dev, shot_Hos, nt*nx*sizeof(float), hipMemcpyHostToDevice);
    }
/*a***********************************Backward*******************************************/
     for(it=nt-1;it>=0;it--)
     { 
     // if(it==0)printf("  B ",is,it);
       /*a#####################a*/
       /*a##  Reconstruction ##a*/
       /*a#####################a*/
        wavefield_bndr<<<((2*nx+2*nz)+511)/512,512>>>(nnx, nnz, nx, nz, npml, it, nt, s_P, s_Q, P_bndr, Q_bndr, false);
        update_vel<<<(nnx*nnz+511)/512, 512>>>(nx,nz,nnx,nnz,npml,dt,dx,dz,
                                               s_u0,s_w0,s_u1,s_w1,s_P,s_Q,coffx1,coffx2,coffz1,coffz2);
        update_stress<<<(nnx*nnz+511)/512, 512>>>(nx,nz,nnx,nnz,dt,dx,dz,s_u1,s_w1,s_P,s_Q,vp,npml,
                                                  s_px1,s_px0,s_pz1,s_pz0,s_qx1,s_qx0,s_qz1,s_qz0,
                                                  acoffx1,acoffx2,acoffz1,acoffz2,delta,epsilon,fs,ds,zs,is,false);
        s_u0=s_u1; s_w0=s_w1; s_px0=s_px1; s_pz0=s_pz1; s_qx0=s_qx1; s_qz0=s_qz1; 

         /*  if((is==1)&&(it%300==0))
            {
	       hipMemcpy(e, s_P, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
              fwrite(e,4L,nnx*nnz,fpsnap);
            }*/
       /*a#####################a*/
       /*a##     Backward    ##a*/
       /*a#####################a*/
        shot_record<<<(nx+511)/512, 512>>>(nnx, nnz, nx, nz, npml, it, nt, g_P, shot_Dev, false);
        shot_record<<<(nx+511)/512, 512>>>(nnx, nnz, nx, nz, npml, it, nt, g_Q, shot_Dev, false);
        update_vel<<<(nnx*nnz+511)/512, 512>>>(nx,nz,nnx,nnz,npml,dt,dx,dz,
                                               g_u0,g_w0,g_u1,g_w1,g_P,g_Q,coffx1,coffx2,coffz1,coffz2);
        update_stress<<<(nnx*nnz+511)/512, 512>>>(nx,nz,nnx,nnz,dt,dx,dz,g_u1,g_w1,g_P,g_Q,vp,npml,
                                                  g_px1,g_px0,g_pz1,g_pz0,g_qx1,g_qx0,g_qz1,g_qz0,
                                                  acoffx1,acoffx2,acoffz1,acoffz2,delta,epsilon,fs,ds,zs,is,false);
        g_u0=g_u1; g_w0=g_w1; g_px0=g_px1; g_pz0=g_pz1; g_qx0=g_qx1; g_qz0=g_qz1; 

        /*   if((is==1)&&(it%300==0))
            {
	       hipMemcpy(e, g_P, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
              fwrite(e,4L,nnx*nnz,fpsnap);
            }*/
        cal_migration<<<(nx*nz+511)/512, 512>>>(nnx, nnz, nz, npml, migration, s_P, g_P);

        Poynting_Adcigs<<<(nx*nz+511)/512, 512>>>(nnz, nx, nz, npml, na, da, adcigs, 
                                                       s_P, s_Q, s_u0, s_w0, g_P, g_Q, g_u0, g_w0);
     }//it loop end

   }//is loop end

   migration_illum<<<(nx*nz+511)/512, 512>>>(nx, nz, npml, migration, illumination);
   adcigs_illum<<<(nx*nz*na+511)/512, 512>>>(nx, nz, na, da, adcigs, illumination);
   /* output multi-shot migration */
   hipMemcpy(e, migration, nz*nx*sizeof(float), hipMemcpyDeviceToHost);
   laplace_filter(1,nz,nx,e,d);
   fwrite(d,sizeof(float),nx*nz,fpmig);
   /* output multi-shot illumination */
   hipMemcpy(e, illumination, nz*nx*sizeof(float), hipMemcpyDeviceToHost);
   fwrite(e,sizeof(float),nx*nz,fpillum);
   /* output multi-shot adcigs */
   hipMemcpy(Atemp, adcigs, nz*nx*na*sizeof(float), hipMemcpyDeviceToHost);
   fwrite(Atemp,sizeof(float),nz*nx*na,fpadcigs);
   /* output adcigs stk migration */
   stk_adcigs(nx,nz,na,Atemp,d);
   fwrite(d,sizeof(float),nx*nz,fpstk);
   /* output smiled adcigs */
   adcigs_smiled(nx,nz,na,dcdp,Atemp);
   fwrite(Atemp,sizeof(float),nz*nx/dcdp*na,fpadcigs2);

   end = clock();
/*********IS Loop end*********/ 		     
   printf("\n---   Complete!!!!!!!!! \n");  
   printf("total %d shots: %f (min)\n", ns, ((float)(end-start))/60.0/CLOCKS_PER_SEC);



/***********close************/ 
     fclose(fpsnap);   fclose(fpobs);  fclose(fpmig);
     fclose(fpillum);  fclose(fpadcigs);fclose(fpstk);
     fclose(fpadcigs2);
/***********free*************/ 
       hipFree(coffx1);       hipFree(coffx2);
       hipFree(coffz1);       hipFree(coffz2);
       hipFree(acoffx1);      hipFree(acoffx2);
       hipFree(acoffz1);      hipFree(acoffz2);

       hipFree(s_u0);           hipFree(s_u1);
       hipFree(s_w0);           hipFree(s_w1);

       hipFree(s_P);            hipFree(s_Q);

       hipFree(s_px0);          hipFree(s_px1);
       hipFree(s_pz0);          hipFree(s_pz1);
       hipFree(s_qx0);          hipFree(s_qx1);
       hipFree(s_qz0);          hipFree(s_qz1);

       hipFree(g_u0);           hipFree(g_u1);
       hipFree(g_w0);           hipFree(g_w1);

       hipFree(g_P);            hipFree(g_Q);

       hipFree(g_px0);          hipFree(g_px1);
       hipFree(g_pz0);          hipFree(g_pz1);
       hipFree(g_qx0);          hipFree(g_qx1);
       hipFree(g_qz0);          hipFree(g_qz1);

       hipFree(shot_Dev);

       hipFree(P_bndr);        hipFree(Q_bndr);

       hipFree(migration); 
       hipFree(illumination);
       hipFree(adcigs);
/***************host free*****************/
	free(v);	free(e);	free(d);
       free(shot_Hos);    free(Atemp);
}

